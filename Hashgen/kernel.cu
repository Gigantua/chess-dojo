﻿
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <iostream>
#include <fstream>
#include "../Core/math.h"
#include "memmap.h"

using namespace std;

const char* bit_filename = "all_bits.dat";

void ensure_bits_file(int bits)
{
    FILE* file;
    file = fopen(bit_filename, "r");
    if (file) {
        fclose(file);
        return;
    }

    file = fopen(bit_filename, "wb");
    uint64_t buf[4096];
    uint64_t* ptr = buf;
    const uint64_t* end = buf + 4096;

    uint64_t total = 0;
    for (int bits = 0; bits <= 8; bits++) {
        total += ncr(64, bits);
    }
    std::cout << 8ull * total << " Bytes\n";

    for (int bits = 0; bits <= 8; bits++)
    {
        Timer t;
        std::cout << "Writing: " << bits << "...";
        uint64_t count = ncr(64, bits);
        uint64_t value = (1ull << bits) - 1;

        for (uint64_t i = 0; i < count; i++)
        {
            *ptr++ = value;
            if (ptr == end) {
                fwrite(buf, sizeof(uint64_t), ptr - buf, file);
                ptr = buf;
            }

            bit_twiddle_permute(value);
        }
        fwrite(buf, sizeof(uint64_t), ptr - buf, file);
        ptr = buf;
        std::cout << t.ms() << "ms\n";
    }
    fclose(file);
    std::cout << "OK";
}


int main()
{
    ensure_bits_file(8);
    memmap m(bit_filename);

}
